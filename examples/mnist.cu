#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>
#include "../milk.h"

using namespace milk;
using namespace milk::factory;

void mnist_reader(std::string fname,
                  std::vector<Data<gpu>>* X,
                  std::vector<Data<gpu>>* Y,
                  uint rows,
                  uint batch_size = 1) {
  MatrixContainer<cpu> X_(Shape2(rows, 784));
  MatrixContainer<cpu> Y_(Shape2(rows, 1));
  std::ifstream in(fname.c_str());
  assert(in.is_open());
  for (uint i=0; i<rows; i++) {
    in >> Y_[i][0];
    for (uint j=0; j<784; j++)
      in >> X_[i][j];
  }
  X_ *= (1./256.);
  paired_shuffle<cpu>({X_, Y_});
  *X = to_data(X_, batch_size);
  *Y = to_data(Y_, batch_size);
}

int main() {
  InitTensorEngine<gpu>(0);
  Data<gpu>::s = NewStream<gpu>();

  uint batch_size = 64;
  std::vector<Data<gpu>> X, Y, Xtest, Ytest;
  mnist_reader("../../data/mnist/mnist_train.txt", &X, &Y, 60000, batch_size);
  mnist_reader("../../data/mnist/mnist_test.txt", &Xtest, &Ytest, 10000, batch_size);

  std::cout << "Data loaded." << std::endl;

  auto ds = datastream(2);
  auto nn = ff(100,nonlin::tanh()) >>
            ff(100,nonlin::tanh()) >>
            ff(100,nonlin::tanh()) >>
            ff(10,nonlin::id());
  auto loss = smax_xent();
  auto all = ds >> nn >> loss;

  nn->set_updater<adam<gpu>>();

  nn->set_lr(1e-3);
  nn->set_la(1e-4);

  trainer<gpu> t(ds, all);
  for (uint ep=0; ep<10; ep++) {
    t.train({&X, &Y});
    std::cout << t.mean_error({&X, &Y}) << "\t";
    std::cout << t.mean_error({&Xtest, &Ytest}) << std::endl;
  }

  ShutdownTensorEngine<gpu>();
  return 0;
}
