#include "hip/hip_runtime.h"
#define Real double
#define MilkDefaultDev cpu
#include "milk.h"

using namespace milk;
using namespace milk::factory;

class Stats {         // small class to keep track of abs/rel diffs and other
  public:             // stuff that might be of interest when gradchecking
    Real max_abs_diff = 0;
    Real max_rel_diff = 0; // TODO

    void accumulate(Real analytic, Real numeric) {
      Real abs = std::abs(analytic - numeric);
      max_abs_diff = std::max(max_abs_diff, abs);
    }

    void accumulate(const Stats& s) {
      max_abs_diff = std::max(max_abs_diff, s.max_abs_diff);
    }

    void print() {
      std::cout << "Max Abs Diff: " << max_abs_diff << std::endl;
    }
};

template<typename xpu, template <typename> class ltype>
Stats check_grad_wrt(std::shared_ptr<ltype<xpu>> l,
                     Data<xpu>*                  W,
                     std::function<void(void)>   init_delta,
                     std::function<Real(void)>   obj_fn,
                     uint                        verbosity=0) {
  Real eps = 1e-4;
  Stats s;

  W->reset_grad();
  l->forward();
  init_delta();
  l->backward();

  Matrix<xpu>& M = (*W)();
  Matrix<xpu>& dM = W->d();

  MatrixContainer<cpu> M_(M.shape_), dM_(dM.shape_);
  Copy(M_, M); Copy(dM_, dM);

  for (uint i=0; i<M.size(0); i++) {
    for (uint j=0; j<M.size(1); j++) {
      as_tensor<xpu>(M[i][j]) += eps;
      l->forward();
      Real up = obj_fn();
      Copy(as_tensor<xpu>(M[i][j]), as_tensor<cpu>(M_[i][j]));
      as_tensor<xpu>(M[i][j]) -= eps;
      l->forward();
      Real down = obj_fn();
      Real numeric_grad = (up - down) / (2*eps);

      if (verbosity > 0)
        std::cout << dM_[i][j] << "\t" << numeric_grad
                  << std::endl;

      s.accumulate(dM_[i][j], numeric_grad);
      Copy(as_tensor<xpu>(M[i][j]), as_tensor<cpu>(M_[i][j]));
    }
  }
  if (verbosity > 0) std::cout << std::endl;
  return s;
}

template <typename xpu, template <typename> class ltype>
void check_grad(std::shared_ptr<ltype<xpu>> l, uint verbosity=0) {
  uint xdim = 4;
  uint T = 5;
  uint bs = 2;

  auto ins = l->dangling_ins();
  std::vector<Data<xpu>> xs(ins.size());

  for (uint i=0; i<ins.size(); i++) {
    auto& x = xs[i];
    x.init(bs*T,xdim);
    mshadow::Random<xpu, Real>(i).SampleUniform(&(x()), -10., 10.);
    x.reset_grad();
    x.batch_size = bs;
    ins[i]->connect_from(x);
  }

  l->forward(); // this is to init weights

  for (auto W : l->params()) (*W)() *= 10;

  auto obj_fn = [&]() { // this will be specialized (mainly for loss layers)
    Real val = 0;
    for (auto& y : l->outs()) val += sqsum((*y)());
    return val + l->loss();
  };
  auto init_delta = [&]() { // this will also be specialized
    for (auto& y : l->outs()) y->d() += 2 * (*y)();
  };

  Stats s;
  for (auto W : l->params())
    s.accumulate(check_grad_wrt(l, W, init_delta, obj_fn, verbosity));
  for (auto& x : xs)
    s.accumulate(check_grad_wrt(l, &x, init_delta, obj_fn, verbosity));
  s.print();
}

template <typename xpu>
void check_grad(std::shared_ptr<layer::smax_xent<xpu>> l, uint verbosity=0) {
  Data<xpu> x, y;
  x.init(2,3); y.init(2,1); y() = 1;
  l->x.connect_from(x); l->y.connect_from(y);
  mshadow::Random<xpu, Real>(0).SampleUniform(&(x()), -10., 10.);

  auto init_delta = []() {}; // no external error
  auto obj_fn = [&]() { return l->loss(); }; // xent loss value is the objective fn

  auto s = check_grad_wrt(l, &x, init_delta, obj_fn, verbosity);
  s.print();
};

template <typename xpu>
void check_grad(std::shared_ptr<layer::cf_smax_xent<xpu>> l, uint verbosity=0) {
  Data<xpu> x1, x2, y;
  x1.init(2,3); x2.init(2,2); y.init(2,1);
  x1.reset_grad(); x2.reset_grad();
  y() = 0; y()[0] += 1; y()[1] += 5;
  l->x1.connect_from(x1);
  l->x2.connect_from(x2);
  l->y.connect_from(y);

  mshadow::Random<xpu, Real>(0).SampleUniform(&(x1()), -10., 10.);
  mshadow::Random<xpu, Real>(1).SampleUniform(&(x2()), -10., 10.);

  auto init_delta = []() {}; // no external error
  auto obj_fn = [&]() { return l->loss(); }; // xent loss value is the objective fn

  auto s = check_grad_wrt(l, &x1, init_delta, obj_fn, verbosity);
  s.accumulate(check_grad_wrt(l, &x2, init_delta, obj_fn, verbosity));
  s.print();
};

template <typename xpu>
void check_grad(std::shared_ptr<layer::sqerr<xpu>> l, uint verbosity=0) {
  Data<xpu> x, y;
  x.init(2,3); y.init(2,3);
  l->x.connect_from(x); l->y.connect_from(y);
  mshadow::Random<xpu, Real>(0).SampleUniform(&(x()), -10., 10.);
  mshadow::Random<xpu, Real>(1).SampleUniform(&(y()), -10., 10.);

  auto init_delta = []() {}; // no external error
  auto obj_fn = [&]() { return l->loss(); }; // xent loss value is the objective fn

  auto s = check_grad_wrt(l, &x, init_delta, obj_fn, verbosity);
  s.print();
};

template <typename xpu>
void check_grad(std::shared_ptr<layer::proj<xpu>> l, uint verbosity=0) {
  Data<xpu> x;
  x.init(4, 1);
  as_tensor<xpu>(x()[0][0]) = 0;
  as_tensor<xpu>(x()[1][0]) = 2;
  as_tensor<xpu>(x()[2][0]) = 0;
  as_tensor<xpu>(x()[3][0]) = 5;
  l->x.connect_from(x);
  l->forward(); // to init weights
  l->W() *= 10;

  auto init_delta = [&]() { l->h.d() += 2 * l->h(); };
  auto obj_fn     = [&]() { return sqsum(l->h()); };
  check_grad_wrt(l, &(l->W), init_delta, obj_fn, verbosity).print();
}

template <typename xpu>
void check_grad(std::shared_ptr<layer::recursive<xpu>> l, uint verbosity=0) {
  Data<xpu> x(10, 2);
  x.batch_size = 2;
  mshadow::Random<xpu, Real>(0).SampleUniform(&(x()), -10., 10.);
  auto dag = std::make_shared<sdag>();
  dag->adj_list = { {{1,0}, {2,1}}, {}, {{3,0}, {4,1}}, {}, {} };
  //dag->labels = { {0, 1}, {}, {0, 1}, {}, {} };
  x.dag = dag;
  assert(l->V.size() > 1);

  l->x.connect_from(x);
  l->forward(); // to init weights

  auto init_delta = [&]() { l->h.d() += 2 * l->h(); };
  auto obj_fn     = [&]() { return sqsum(l->h()); };
  check_grad_wrt(l, &(l->W), init_delta, obj_fn, verbosity).print();
}

#define CHECK_GRAD(layer)                                    \
std::cout << "Checking " << #layer << std::endl;             \
check_grad(layer, verbosity);                                \
std::cout << std::endl;                                      \

int main(int argc, char** argv) {
  InitTensorEngine<MilkDefaultDev>();
  uint verbosity = 0;
  if (argc > 1) verbosity = stoi(std::string(argv[1]));

  CHECK_GRAD( ff(3) )
  CHECK_GRAD( ff(3) >> ff(2) )
  CHECK_GRAD( smax_xent() )
  CHECK_GRAD( sqerr() )
  CHECK_GRAD( recurrent(3) )
  CHECK_GRAD( recurrent(3,reverse) )
  CHECK_GRAD( cat() )
  CHECK_GRAD( cast() )
  CHECK_GRAD( tail() )
  CHECK_GRAD( tailcast() )
  CHECK_GRAD( (ff(3), ff(2)) )
  CHECK_GRAD( cast()
              >> (recurrent(3), recurrent(2,reverse))
              >> cat() )
  //CHECK_GRAD( proj(3, 5) )

  CHECK_GRAD( lstm(3) )
  CHECK_GRAD( cf_smax_xent() )

  CHECK_GRAD( timewise(lstm(3)) )
  CHECK_GRAD( timewise(lstm(3) >> lstm(2)) )
  CHECK_GRAD( timewise(ff(3)) )

  CHECK_GRAD( recursive(3,2) )

  ShutdownTensorEngine<MilkDefaultDev>();

  return 0;
}
